#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

#define BLOCKS  1
#define THREADS 128
__global__ void addNums(float* output, float* x, float* y, int iter) {
    // Add several value in this thread
	int step = (iter + THREADS - 1) / THREADS;
	int base = threadIdx.x * step;	
	float result = 0;
	for(unsigned int i= 0; i < step; ++i) {
		int idx = base + i;
		if(idx >= iter) break;
		//result += x[idx] * y[idx];	
		result = __fmaf_rd(x[idx], y[idx], result);	
	}

	// Store result in thread share memory
	__shared__ float tmp[THREADS];
	tmp[threadIdx.x] = result;

	// Wait until all thread have finished their task
	__syncthreads();

    // Accumulate result of all thread if we are thread #0
	if(threadIdx.x == 0) {
		result = 0;
		for(int i= 0; i < THREADS; ++i) {
			result += tmp[i];
		}
		*output = result;
	}
}

#define num_iters 12000000
int main() {
	float *x;
	float *y;
	float *output;

	srand(10);

    // Allocate shared memory
	hipMallocManaged(&x, num_iters*sizeof(float));
	hipMallocManaged(&y, num_iters*sizeof(float));
	hipMallocManaged(&output, sizeof(float));

    // Initialise the input arrays
	*output = 0;
	for(unsigned int i= 0; i < num_iters; ++i) {
		x[i]= 1;
		y[i]= 1;
	}

	std::cout << "Hello World! this is CUDA sample code" << std::endl;
    //int deviceCount;
    //cudaGetDeviceCount(&deviceCount);
	//std::cout << "Device count: " << deviceCount << std::endl;

    // Compute the sum of arrays
	addNums<<<BLOCKS,THREADS>>>(output, x, y, num_iters);
	hipDeviceSynchronize();

	std::cout << "Sum is: " << *output << std::endl;

    // Release the shared memory
	hipFree(x);
	hipFree(y);
	hipFree(output);

	std::cout << "Code Execution Completed" << std::endl;

	return 0;
}

